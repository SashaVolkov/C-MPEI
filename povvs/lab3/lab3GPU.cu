
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <math.h>

__global__ void Add( int n, float *A, float *B, float *C, float S1, float S2, int steps) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < n){
		for (int i = 0; i < steps; i++){
			C[steps*idx+i]=(powf(S1,3.0)- 3*B[steps*idx+i])/(A[steps*idx+i] + S2); //powf(S1,3.0)
		}
	}
}


int main ( int argc, char * argv [] ) {

	if (argc != 4){
		printf("To few args\n");
		return 1;
	}

	int n = atoi(argv[1]);
	printf("n = %d\n", n);

	int steps;
	float *hA, *hB, *hC;
	// float hS1, hS2;
	float *devA, *devB, *devC;
	float devS1, devS2, elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	elapsedTime = 0.0f;
	hipEventRecord(start, 0);

	hA = (float *) malloc(n*sizeof(float));
	hB = (float *) malloc(n*sizeof(float));
	hC = (float *) malloc(n*sizeof(float));


	hipMalloc ( &devA, n*sizeof(float));
	hipMalloc ( &devB, n*sizeof(float));
	hipMalloc ( &devC, n*sizeof(float));

	// cudaMalloc (&devS1, sizeof(float));
	// cudaMalloc (&devS2, sizeof(float));

	for (int i = 0; i < n; i++) {
		hA[i] = i;
		hB[i] = n-i;
	}
	devS1 = 15.4;
	devS2 = 4.4;


	// blocks=4; blocksize=64;
	dim3 block(atoi(argv[3]));
	dim3 grid(atoi(argv[2]));
	steps=(int)n/(atoi(argv[3])*atoi(argv[2]));

	// dim3 block(512);
	// dim3 grid((n-1)/512 + 1);

	printf("block = %d, grid = %d, threads = %d, steps = %d\n", block.x, grid.x, block.x*grid.x, steps);




	hipMemcpy ( devA, hA, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy ( devB, hB, n*sizeof(float), hipMemcpyHostToDevice);


	Add<<<grid, block>>> ( n, devA, devB, devC, devS1, devS2, steps);

	hipMemcpy ( hC, devC, n*sizeof(float), hipMemcpyDeviceToHost );


	for (int i = n-10; i < n; i++) printf("devC[%d] = %.3f\n", i, hC[i]);

	hipFree (devC);
	hipFree (devA);
	hipFree (devB);
	free(hA);
	free(hB);
	free(hC);



	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("\n\ntime = %.5f millisec\n", elapsedTime);

	return 0;
}

// Yi =(S1^m - 3Bi)/(Ai + S2)