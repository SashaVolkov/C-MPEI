
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <math.h>

__global__ void Add( int n, float *A, float *B, float *C, float S1, float S2) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < n)
 C[idx]=(powf(S1,3.0)- 3*B[idx])/(A[idx] + S2); //powf(S1,3.0)
}


int main ( int argc, char * argv [] ) {

	if (argc != 2) return 1;

	int n = atoi(argv[1]);
	printf("n = %d\n", n);

	// int i, steps;
	float *hA, *hB, *hC;
	// float hS1, hS2;
	float *devA, *devB, *devC;
	float devS1, devS2, elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	elapsedTime = 0.0f;
	hipEventRecord(start, 0);

	hA = (float *) malloc(n*sizeof(float));
	hB = (float *) malloc(n*sizeof(float));
	hC = (float *) malloc(n*sizeof(float));


	hipMalloc ( &devA, n*sizeof(float));
	hipMalloc ( &devB, n*sizeof(float));
	hipMalloc ( &devC, n*sizeof(float));

	// cudaMalloc (&devS1, sizeof(float));
	// cudaMalloc (&devS2, sizeof(float));

	for (int i = 0; i < n; i++) {
		hA[i] = i;
		hB[i] = n-i;
	}
	devS1 = 15.4;
	devS2 = 4.4;


	// blocks=4; blocksize=64;
	// steps=(int)n/(blocks*blocksize);
	// dim3 block(atoi(argv[3]));
	// dim3 grid(atoi(argv[2]));

	dim3 block(512);
	dim3 grid((n-1)/512 + 1);

	printf("block = %d, grid = %d, threads = %d\n", block.x, grid.x, block.x*grid.x);




	hipMemcpy ( devA, hA, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy ( devB, hB, n*sizeof(float), hipMemcpyHostToDevice);


	Add<<<grid, block>>> ( n, devA, devB, devC, devS1, devS2);

	hipMemcpy ( hC, devC, n*sizeof(float), hipMemcpyDeviceToHost );


	for (int i = n-10; i < n; i++) printf("devC[%d] = %.3f\n", i, hC[i]);

	hipFree (devC);
	hipFree (devA);
	hipFree (devB);
	free(hA);
	free(hB);
	free(hC);



	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("\n\ntime = %.5f millisec\n", elapsedTime);

	return 0;
}

// Yi =(S1^m - 3Bi)/(Ai + S2)