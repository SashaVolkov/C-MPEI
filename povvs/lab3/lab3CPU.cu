
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <math.h>

int main ( int argc, char * argv [] ) {

	if (argc != 2) return 1;

	int n = atoi(argv[1]);
	printf("n = %d\n", n);

	// int i, steps;
	float *A, *B, *C;
	float S1, S2, elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	elapsedTime = 0.0f;
	hipEventRecord(start, 0);

	A = (float *) malloc(n*sizeof(float));
	B = (float *) malloc(n*sizeof(float));
	C = (float *) malloc(n*sizeof(float));


	for (int i = 0; i < n; i++) {
		A[i] = i;
		B[i] = n-i;
	}
	S1 = 15.4;
	S2 = 4.4;


	for (int i = 0; i < n; i++) C[i]=(powf(S1,3.0)- 3*B[i])/(A[i] + S2);


	for (int i = n-10; i < n; i++) printf("devC[%d] = %.3f\n", i, C[i]);

	free(A);
	free(B);
	free(C);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("\n\ntime = %.5f millisec\n", elapsedTime);

	return 0;
}

// Yi =(S1^m - 3Bi)/(Ai + S2)