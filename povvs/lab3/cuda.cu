
#include <hip/hip_runtime.h>
# include <stdio.h>

// # define n 4096

__global__ void Add( int n, float *A, float *B, float *C)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if(idx < n)
 C[idx]=A[idx]+B[idx];
}


int main ( int argc, char * argv [] ) {

	if (argc != 2) return 1;

	int n = atoi(argv[1]);
	printf("n = %d\n", n);

	// int i, steps;
	float *hA, *hB, *hC;
	float * devA, *devB, *devC;

	hA = (float *) malloc(n*sizeof(float));
	hB = (float *) malloc(n*sizeof(float));
	hC = (float *) malloc(n*sizeof(float));


	hipMalloc ( &devA, n*sizeof(float));
	hipMalloc ( &devB, n*sizeof(float));
	hipMalloc ( &devC, n*sizeof(float));

	for (int i = 0; i < n; i++) {
		hA[i] = i;
		hB[i] = n-i;
	}

	// blocks=4; blocksize=64;
	// steps=(int)n/(blocks*blocksize);
	dim3 block(512);
	dim3 grid((n-1)/512 + 1);
	printf("block = %d, grid = %d, threads = %d\n", block.x, grid.x, block.x*grid.x);

	hipMemcpy ( devA, hA, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy ( devB, hB, n*sizeof(float), hipMemcpyHostToDevice);

	Add<<<grid, block>>> ( n, devA, devB, devC);

	hipMemcpy ( hC, devC, n*sizeof(float), hipMemcpyDeviceToHost );


	for (int i = n-10; i < n; i++) printf("devC[%d] = %.5f\n", i, hC[i]);

	hipFree (devC);
	hipFree (devA);
	hipFree (devB);
	free(hA);
	free(hB);
	free(hC);

	return 0;
}