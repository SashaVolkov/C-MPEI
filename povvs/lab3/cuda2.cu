int main ( int argc, char * argv [] ){

	int i,blocks,blocksize,steps;
	float vect1[N],vect2[N];
	FILE *f;
	float * devA, *devB, elapsedTime;
	hipEvent_t start, stop; //Идентификаторы событий

	hipEventCreate(&start); //Инициализация события start
	hipEventCreate(&stop); //Инициализация события stop

	for (i=0; i<N; i++) { vect1[i]=i; vect2[i]=i;}

	hipMalloc ( (void**)&devA, N * sizeof ( float ) ) ;
	hipMalloc ( (void**)&devB, N * sizeof ( float ) ) ;

	blocks=16; blocksize=128;
	steps=(int)N/(blocks*blocksize);

	hipEventRecord(start,0); // Фиксация события start
	hipMemcpy ( devA, vect1, N * sizeof ( float ), hipMemcpyHostToDevice);
	hipMemcpy ( devB, vect2, N*sizeof ( float ), hipMemcpyHostToDevice);

	FUN_KERNEL<<<blocks,blocksize>>> ( devA, devB, steps);

	hipMemcpy ( vect1, devA, N * sizeof ( float ), hipMemcpyDeviceToHost );
	hipFree ( devA );
	hipEventRecord(stop,0); // Фиксация события stop
	hipEventSynchronize(stop); // Синхронизация host и device по событию stop
	// Определение времени (в миллисекундах) между событиями start и stop
	hipEventElapsedTime(&elapsedTime,start,stop);

	printf("time1 = %f\n", elapsedTime); //Вывод времени
	for (i = 0; i < N; i++) printf("vect1[%d] = %.5f\n", i, vect1[i]);
		
	return 0;
}
